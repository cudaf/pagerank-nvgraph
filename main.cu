#include <vector>
#include <stdio.h>
#include "src/main.h"

using namespace std;




template <class G>
void runPageRank(G& x, float p=0.85f, float E=1e-6f) {
  nvgraphHandle_t     h;
  nvgraphGraphDescr_t g;
  struct nvgraphCSCTopology32I_st csc;
  vector<hipDataType> vtype {HIP_R_32F, HIP_R_32F};
  vector<hipDataType> etype {HIP_R_32F};
  vector<float> ranks(x.order());
  auto vfrom = x.sourceOffsets();
  auto efrom = x.destinationIndices();
  auto vdata = x.vertexData();
  auto edata = x.edgeData();

  TRY( nvgraphCreate(&h) );
  TRY( nvgraphCreateGraphDescr(h, &g) );

  csc.nvertices = x.order();
  csc.nedges    = x.size();
  csc.destination_offsets = vfrom.data();
  csc.source_indices      = efrom.data();
  TRY( nvgraphSetGraphStructure(h, g, &csc, NVGRAPH_CSC_32) );

  TRY( nvgraphAllocateVertexData(h, g, vtype.size(), vtype.data()) );
  TRY( nvgraphAllocateEdgeData(h, g, etype.size(), etype.data()) );
  TRY( nvgraphSetVertexData(h, g, vdata.data(), 0) );
  TRY( nvgraphSetVertexData(h, g, ranks.data(), 1) );
  TRY( nvgraphSetEdgeData(h, g, edata.data(), 0) );

  float t = measureDuration([&]() { TRY( nvgraphPagerank(h, g, 0, &p, 0, 0, 1, E, 0) ); });
  TRY( nvgraphGetVertexData(h, g, ranks.data(), 1) );
  printf("[%07.1f ms] pageRank\n", t); print(ranks);

  TRY( nvgraphDestroyGraphDescr(h, g) );
  TRY( nvgraphDestroy(h) );
}


int main(int argc, char **argv) {
  printf("Loading graph ...\n");
  DiGraph<> g;
  readMtx(argv[1], g);
  print(g);

  DiGraph<int, float, float> h;
  transposeForNvgraph(g, h);
  print(h);

  runPageRank(h);
  return 0;
}
